#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>
#include <fstream>  
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <time.h> 
#include <string>
#include "struct_cu.h"
#include "constants.h"
//#include "buffer.h"
using namespace std;
// assume block size equal population size

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
	int tid = threadIdx.x;
	hiprand_init(seed, tid, 0, state + tid);
}

__device__ float fitness(M_args deviceParameter, M_args_Tset deviceParameter_Tset,float tau) {
    float result = 0;
	//printf("%d_a",deviceParameter.spike_data_num);
	//printf("%d_b", deviceParameter_Tset.length);
	for (size_t i = 0; i<deviceParameter_Tset.length; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
	{
		result += expf(-fabsf(deviceParameter_Tset.spike_TestData[i] - deviceParameter_Tset.spike_TestData[j])*1.0 / tau);
		//printf("%f_3 ", result);
		// ++curPos;
	}

	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter.spike_data_num; ++j)
		{
			result += expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter.spike_data[j])*1.0 / tau);
			//printf("%f_2 ", result);
			// ++curPos;
		}
	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
		{
			//printf("%f_c ", deviceParameter.spike_data[i]);
			//printf("%f_d ", deviceParameter_Tset.spike_TestData[j]);
			result -= 2*expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter_Tset.spike_TestData[j])*1.0 / tau);
			//printf("%f_1 ", result);
			// ++curPos;
		}
    //printf("%f_4 ", result);
    return result;
}


__global__ void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau, int genindex, int MaxGeneration, M_args_Bound *deviceParameter_Bound, const int POPULATION_SIZE, float crossver,float mutations) {
	// we first have to calculate the score for the first half of threads
	//const float *curPos = sharedPopulation[tid];
	__shared__ float sharedPopulation[THREADS_PER_BLOCK*2][VAR_NUMBER];
	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
	//__shared__ float **sharedPopulation = new float*[THREADS_PER_BLOCK * 2];
	__shared__ float sharedScore[THREADS_PER_BLOCK*2];
	//const float SIGN[2] = { -1.0f, 1.0f };
	//const float MULT[2] = { 1.0f, 0.0f };

	const int gid = blockDim.x * blockIdx.x + threadIdx.x + THREADS_PER_BLOCK*blockIdx.x;
	const int tid = threadIdx.x;
	printf("i_%f ", crossver);
	// loading initial random population into shared memory
	if ((gid + THREADS_PER_BLOCK)< POPULATION_SIZE) {
		for (int i = 0; i < VAR_NUMBER; ++i){
			__syncthreads();

			sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
			sharedPopulation[tid + THREADS_PER_BLOCK][i] = population[(gid + THREADS_PER_BLOCK) * VAR_NUMBER + i];
		}
		M_args_Tset curPos = deviceParameter_Tset[gid];
		sharedScore[tid] = fitness(deviceParameter, curPos, tau);
		M_args_Tset curPos_b = deviceParameter_Tset[(gid + THREADS_PER_BLOCK)];
		sharedScore[tid + THREADS_PER_BLOCK] = fitness(deviceParameter, curPos_b, tau);
		__syncthreads();
		}

	//sharedScore[tid + THREADS_PER_BLOCK] = 123123.0;
	hiprandState &localState = randomStates[(tid*genindex)%THREADS_PER_BLOCK];
	__syncthreads();


	//__syncthreads();
	if (genindex <= MaxGeneration && (gid) < POPULATION_SIZE)
	{// selection
		// first half of threads writes best individual into its position
			if (sharedScore[tid] > sharedScore[tid + THREADS_PER_BLOCK]) {
				for (int i = 0; i < VAR_NUMBER; ++i)
					sharedPopulation[tid][i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
				sharedScore[tid] = sharedScore[tid + THREADS_PER_BLOCK];
			}
		
		__syncthreads();
		////int temp_size = THREADS_PER_BLOCK*(1 - crossver);
		////if (tid<temp_size && sharedScore[tid] > sharedScore[tid + temp_size]) {
		////	for (int i = 0; i < VAR_NUMBER; ++i)
		////		sharedPopulation[tid][i] = sharedPopulation[tid + temp_size][i];
		////	sharedScore[tid] = sharedScore[tid + temp_size];
		////}
		__syncthreads();
		// crossover
		////printf("i_%f ", crossver);
		////printf("gi_%f ", hiprand_uniform(&localState));
		if (hiprand_uniform(&localState) < crossver) {
			hiprandState &localState1 = randomStates[tid * 3];
			hiprandState &localState2 = randomStates[tid * 4];
			const int first = hiprand_uniform(&localState1) * THREADS_PER_BLOCK;
			const int second = hiprand_uniform(&localState2) * THREADS_PER_BLOCK;
			const float weight = hiprand_uniform(&localState1);
			for (int i = 0; i < VAR_NUMBER; ++i) {
				////const float Temp_weight = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
				//////printf("i_%f ", Temp_weight);
				////if (Temp_weight>deviceParameter_Bound[i].g[1] || Temp_weight < deviceParameter_Bound[i].g[0])
				////	sharedPopulation[tid + THREADS_PER_BLOCK][i] = deviceParameter_Bound[i].g[0] + weight*(deviceParameter_Bound[i].g[1] - deviceParameter_Bound[i].g[0]);
				////else
				////	sharedPopulation[tid + THREADS_PER_BLOCK][i] = Temp_weight;
				//printf("i_%f %f", deviceParameter_Bound[i].g[0], deviceParameter_Bound[i].g[1]);
				//printf("i_%f ", Temp_weight);
				sharedPopulation[tid + THREADS_PER_BLOCK][i] = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
				
			}
		}
		__syncthreads();

		// mutations on second half of population
		hiprandState &localState3= randomStates[tid*5];
		if (hiprand_uniform(&localState3) < mutations) {
			//const float order = (hiprand_uniform(&localState) - mutations*1.0/2);
			float guass = hiprand_normal(&localState3);
			for (int i = 0; i < VAR_NUMBER; ++i) {
				////const float mult = MULT[order < 0.0f];
				////if (sharedPopulation[tid + THREADS_PER_BLOCK][i]>deviceParameter_Bound[i].g[1] || sharedPopulation[tid + THREADS_PER_BLOCK][i] < deviceParameter_Bound[i].g[0])
				sharedPopulation[tid + THREADS_PER_BLOCK][i] = (deviceParameter_Bound[i].g[0] + deviceParameter_Bound[i].g[1]) / 2 + guass;
			}
		}
		__syncthreads();
		const int third = hiprand_uniform(&localState)*THREADS_PER_BLOCK;
		//////sharing a part of population with others
		////if (hiprand_uniform(&localState)<0.1) {
		////	for (int i = 0; i < VAR_NUMBER; ++i)
		////		population[gid * VAR_NUMBER + i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
		////}

		// take some best individuals from neighbour
		if ((blockIdx.x + third) % 3 == 0) {
			if (hiprand_uniform(&localState) < 0.11) {
				const int anotherBlock = hiprand_uniform(&localState) * (POPULATION_SIZE / THREADS_PER_BLOCK);
				const int ngid = blockDim.x * anotherBlock + threadIdx.x;
				for (int i = 0; i < VAR_NUMBER; ++i)
					sharedPopulation[tid][i] = population[ngid * VAR_NUMBER + i];
				//sharedScore[tid] = fitness(sharedPopulation[tid], deviceParameter);
				//sharedScore[tid]=fitness(deviceParameter, curPos_b, tau);
			}
		}
	}
	__syncthreads();
	////// output current population back
	if ((gid ) < POPULATION_SIZE) {
		for (int i = 0; i < VAR_NUMBER; ++i)
			{
				population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];
				population[(gid + THREADS_PER_BLOCK) * VAR_NUMBER + i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
				printf("%f ", population[gid * VAR_NUMBER + i]);
			}
	
	if (genindex <= MaxGeneration)
		{
			score[gid].score = sharedScore[tid];
		}
	}
}


void printFinalPopulation(float* population, const ScoreWithId* deviceScore, const int POPULATION_SIZE) {
	////float **population = new float *[POPULATION_SIZE];
	////for (int i = 0; i < POPULATION_SIZE; i++)
	////	population[i] = new float[VAR_NUMBER];
	////cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId *score = new ScoreWithId[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	///std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);

	for (int i = 0; i<1; ++i) {
		std::cout <<"gen_index"<< std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i=0; i<VAR_NUMBER; i++) {
			for (int u=0; u<POPULATION_SIZE; ++u) {
				std::cout << std::setw(15) << population[u*VAR_NUMBER + i] << ' ';
			}
			std::cout << std::endl;
		}
	std::cout << "Score: " << std::endl;
	for (int i = 0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
	//delete population;
}
extern "C" float solveGPU(M_args &Parameter_, double Mtime, double tempVB, double TimeStep, vector<float> m_I, int FlagParameter[], M_args_Bound Parameter_Bound[], int MaxGeneration, float gL, float C, const int POPULATION_SIZE, float crossver, float mutations, stringstream &strResult) {
    cudasafe(hipSetDevice(0), "Could not set device 0");
	float ans = 0;
	clock_t start, finish, startMy, finishMy;
	float tau = 12;
	//M_args *IndexParameter_ = new M_args[MaxGeneration];
	//IndexParameter_ = 0;
	//IndexParameter_[0] = Parameter_;
	/////////////////////////////////////
	////cout << Parameter_.spike_data_num << endl;
	////for (int i = 0; i < Parameter_.spike_data_num; i++)
	////	cout << Parameter_.spike_data[i] << " ";
	////cout << endl;
	////cout << mutations << " " << crossver<<endl;
	////std::cout << std::setw(15) << Parameter_Bound[0].g[0] << " " << Parameter_Bound[0].g[1] << " " << tempVB << endl;
	std::cout << FlagParameter[0] << ' ' << FlagParameter[1] << ' ' << FlagParameter[2] << ' ' << FlagParameter[3] << ' ' << FlagParameter[4] << endl;
	std::cout << Parameter_Bound[0].g[0] << ' ' << Parameter_Bound[1].g[0] << ' ' << Parameter_Bound[2].g[0] <<' ' << Parameter_Bound[3].g[0]<<endl;
	//////////////////////////////////////
	float *population = new float[POPULATION_SIZE*VAR_NUMBER];
	#pragma omp parallel for num_threads(4)
	for (int i=0; i<POPULATION_SIZE; i++) {
		for (int j=0; j<VAR_NUMBER; j++) {
			if (FlagParameter[j] == 0)
			{
				population[i*VAR_NUMBER + j] = 0;
			}
			else
			{
				population[i*VAR_NUMBER + j] = (float_random(Parameter_Bound[j]));
				//std::cout << std::setw(15) << population[i*VAR_NUMBER + j] << ' ';
			}
		}
		//std::cout << endl;
	}

	M_args_Tset *Parameter_Tset=new M_args_Tset[POPULATION_SIZE];

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	M_args_Tset *deviceParameter_Tset = 0;
	
	hiprandState* randomStates;
	M_args deviceParameter_;
	//deviceParameter_.current_data_num = Parameter_.current_data_num;
	deviceParameter_.spike_data_num = Parameter_.spike_data_num;
	M_args_Bound *deviceParameter_Bound = 0;
	//int DataLength = getArrayLen(Parameter_.spike_data);


	//int DataLengthC = getArrayLen(Parameter_.current_data);
	//Parameter_.length = DataLength;
	cudasafe(hipMalloc((void **)&deviceParameter_Bound, VAR_NUMBER * sizeof(M_args_Bound)), "Could not allocate memory for deviceParameter_Bound");

	cudasafe(hipMalloc(&randomStates, THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceParameter_Tset, 2*POPULATION_SIZE * sizeof (M_args_Tset)), "Could not allocate memory for deviceParameter_Tset");
	//cudasafe(hipMalloc((void **)&deviceParameter_.current_data, Parameter_.current_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	cudasafe(hipMalloc((void **)&deviceParameter_.spike_data, Parameter_.spike_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_, sizeof(M_args)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_.spike_TestData, DataLength*sizeof(float)), "Could not allocate memory for deviceParameter_");

	///cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");
	//cudasafe(hipMemcpy(deviceParameter_.current_data, Parameter_.current_data, Parameter_.current_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_current_data to device");
	cudasafe(hipMemcpy(deviceParameter_.spike_data, Parameter_.spike_data, Parameter_.spike_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_spike_data to device");

	//cudasafe(hipMemcpy(deviceParameter_.spike_TestData, Parameter_.spike_TestData, DataLength*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_ to device");
	for (int kb = 0; kb<VAR_NUMBER;kb++)
		cudasafe(hipMemcpy(&deviceParameter_Bound[kb], &Parameter_Bound[kb], 2 * sizeof(float),hipMemcpyHostToDevice), "Could not allocate memory for deviceParameter_Bound");


	// invoking random init
	randomInit<<<1, THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	
	//__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
    //for (int i=0; i<1115; i++) {
	//void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau)
	for (int k = 1; k <=MaxGeneration; k++) 
	{
		m_gen = k;
		hDll = LoadLibrary("CHdll.dll");
		startMy = clock();
		start = clock();
		int spike_length = int(Mtime / TimeStep);
		float *temp_data = new float[spike_length*POPULATION_SIZE];
		HH_return(population, VAR_NUMBER, Mtime, tempVB, TimeStep, m_I, FlagParameter, gL, C, temp_data,POPULATION_SIZE);
		FreeLibrary(hDll);
		//Parameter_Tset[j].spike_TestData = HH_SpikeTime(temp_data, Mtime, TimeStep, Parameter_Tset[j].length, Parameter_.spike_data_num);
		/////////////
		////for (int j = 0; j < POPULATION_SIZE; j++)
		////	for (int i = 0; i < spike_length; i++)
		////		cout << temp_data[j*spike_length+i] << endl;
		//////////////////
		//#pragma omp parallel for
		for (int j = 0; j < POPULATION_SIZE; j++)
		{
			float *temp_spike_TestData;
			vector<float>mdata_tmp = HH_SpikeTime(&temp_data[j*spike_length], Mtime, TimeStep, Parameter_Tset[j].length, Parameter_.spike_data_num);
			Parameter_Tset[j].spike_TestData = new float[mdata_tmp.size()];
			convert_data_two(mdata_tmp, Parameter_Tset[j].spike_TestData);
			hipMalloc(&temp_spike_TestData, Parameter_Tset[j].length*sizeof(float));
			//std::cout << Parameter_Tset[j].length << std::endl;
			cudasafe(hipMemcpy(&deviceParameter_Tset[j], &Parameter_Tset[j], sizeof (M_args_Tset), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset1 to device");
			cudasafe(hipMemcpy(temp_spike_TestData, Parameter_Tset[j].spike_TestData, (Parameter_Tset[j].length*sizeof(float)), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData2 to device");
			cudasafe(hipMemcpy(&deviceParameter_Tset[j].spike_TestData, &temp_spike_TestData, sizeof(float*), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData to device");
			cudasafe(hipFree(temp_spike_TestData), "Could not free temp_spike_TestData");
			delete Parameter_Tset[j].spike_TestData;
			//cout << j << endl;
		}
		delete temp_data;
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time1 " << duration << endl;
		ScoreWithId *deviceScore = 0;
		cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");
		cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");

		start = clock();
		int A = 0;

		GAKernel_GenEach << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau, k, MaxGeneration, deviceParameter_Bound, POPULATION_SIZE, crossver, mutations);
		//delete[]population;
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time2 " << duration << endl;
		//float *population = new float[POPULATION_SIZE * VAR_NUMBER];
		cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");
		cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
		//printf("%d_1111\n", k);
		//printFinalPopulation(population, deviceScore, POPULATION_SIZE);
		//printf("%d_2222\n", k);
		//while (MFlage)
		/////////////////////exit-code
		if (myexit)
		{
			cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");

			cudasafe(hipFree(randomStates), "Could not free randomStates");
			cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
			cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
			delete Parameter_Tset;
			delete population;
			return 0;
		}
		///////////////////////
		cudasafe(hipGetLastError(), "Could not invoke GAKernel");
		cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling GAKernel");

		//printPopulation(devicePopulation, deviceScore);
		//}


		mtx.lock();
		start = clock();
		ans = FinallResult(k, tau, population, Mtime, tempVB, TimeStep, m_I, Parameter_, FlagParameter, gL, C, Parameter_Bound, POPULATION_SIZE, strResult);
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time3 " << duration << endl;
		//MFlage = 3;
		mtx.unlock();
		finishMy = clock();
		
		durationMy = (double)(finishMy - startMy) / CLOCKS_PER_SEC;
		cout << "timeAll " << durationMy << endl;
		/////////////////////exit-code
		if (myexit)
		{
			cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");

			cudasafe(hipFree(randomStates), "Could not free randomStates");
			cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
			cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
			delete Parameter_Tset;
			delete population;
			return 0;
		}
		
		//////////////////////
		////GAKernel_gen << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, sharedPopulation, sharedScore, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau);
	}
	
	//ans = FinallResult(tau, population, Mtime, tempVB, TimeStep, m_I, Parameter_, FlagParameter, gL, C, Parameter_Bound, POPULATION_SIZE, strResult);
	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");

	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
	cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
	
	delete Parameter_Tset;
	//////////////////////
	delete population;
	myexit = true;
	return ans;
}











////////////////////////////////////////////////////////////\\gpu_HH
/////////////////////////////////////////////////////////////\\
///////////////////////////////////////////////////////////////
/////////////////////////
///////////////////////////////////////////////Ca
////class CCa
////{
////protected:
////	double g_Ca;
////	double m_c;
////public:
////	__device__ CCa(double gca, double n);
////	__device__ double Compute_n(double tempV, double TimeStep);
////	__device__ double Compute_KI(double tempV, double TimeStep);
////	__device__ double Iterative(double tmp_n, double an, double bn, double TimeStep);
////	__device__ CCa();
////	__device__ ~CCa();
////};
////__device__ CCa::CCa(double gca, double n)
////{
////	g_Ca = gca;
////	m_c = n;
////}
////__device__ double CCa::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double CCa::Compute_n(double tempV, double TimeStep)
////{
////	double an = 0.3*(tempV + 13) / (1 - exp(-(tempV + 13)*1.0 / 10));
////	double bn = 10.0 * exp(-(tempV + 38) *1.0 / 18);
////	m_c = Iterative(m_c, an, bn, TimeStep);
////	return m_c;
////}
////__device__ double CCa::Compute_KI(double tempV, double TimeStep)
////{
////	double tmp_n = Compute_n(tempV, TimeStep);
////	double Ca_I = g_Ca * pow(tmp_n, 3)*(tempV - 120);
////	return Ca_I;
////}
////__device__ CCa::CCa()
////{}
////__device__ CCa::~CCa()
////{
////}
////
/////////////////////////////////////////////////////////K
////class CK
////{
////protected:
////	double g_K;
////	double m_Kn;
////public:
////	__device__ CK(double gk, double n);
////	__device__ double Compute_n(double tempV, double TimeStep);
////	__device__ double Compute_KI(double tempV, double TimeStep);
////	__device__ double Iterative(double tmp_n, double an, double bn, double TimeStep);
////	__device__ CK();
////	__device__ ~CK();
////};
////__device__ CK::CK(double gk, double n)
////{
////	g_K = gk;
////	m_Kn = n;
////}
////__device__ double CK::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double CK::Compute_n(double tempV, double TimeStep)
////{
////	double an = 0.01*(tempV + 55) / (1 - exp(-0.1*(tempV + 55)));
////	double bn = 0.125*exp(-(tempV + 65) *1.0 / 80);
////	m_Kn = Iterative(m_Kn, an, bn, TimeStep);
////	return m_Kn;
////}
////__device__ double CK::Compute_KI(double tempV, double TimeStep)
////{
////	double tmp_n = Compute_n(tempV, TimeStep);
////	double K_I = g_K * pow(tmp_n, 4)*(tempV + 77);
////	return K_I;
////}
////__device__ CK::CK()
////{}
////__device__ CK::~CK()
////{
////}
////////////////////////////////////////////////////////KM
////class CKM
////{
////protected:
////	double g_KM;
////	double m_z;
////public:
////	__device__ CKM(double gk, double n);
////	__device__ double Compute_n(double tempV, double TimeStep);
////	__device__ double Compute_KI(double tempV, double TimeStep);
////	__device__ double Iterative(double tmp_n, double an, double bn, double TimeStep);
////	__device__ CKM();
////	__device__ ~CKM();
////};
////
////__device__ CKM::CKM(double gk, double n)
////{
////	g_KM = gk;
////	m_z = n;
////}
////__device__ double CKM::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double CKM::Compute_n(double tempV, double TimeStep)
////{
////	double an = 1.0 / (1 + exp(-0.2*(tempV + 39))) / 75;
////	double bn = 1.0 / 75 - an;
////	m_z = Iterative(m_z, an, bn, TimeStep);
////	return m_z;
////}
////__device__ double CKM::Compute_KI(double tempV, double TimeStep)
////{
////	double tmp_n = Compute_n(tempV, TimeStep);
////	double K_I = g_KM * pow(tmp_n, 1)*(tempV + 77);
////	return K_I;
////}
////__device__ CKM::CKM()
////{}
////__device__ CKM::~CKM()
////{
////}
////////////////////////////////////////////////////////Kv
////class CKv
////{
////protected:
////	double g_Kv;
////	double m_p;
////public:
////	__device__ CKv(double gk, double n);
////	__device__ double Compute_n(double tempV, double TimeStep);
////	__device__ double Compute_KI(double tempV, double TimeStep);
////	__device__ double Iterative(double tmp_n, double an, double bn, double TimeStep);
////	__device__ CKv();
////	__device__ ~CKv();
////};
////
////__device__ CKv::CKv(double gk, double n)
////{
////	g_Kv = gk;
////	m_p = n;
////}
////__device__ double CKv::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double CKv::Compute_n(double tempV, double TimeStep)
////{
////	double an = (tempV - 95)*1.0 / (1 - exp(-(tempV - 95) / 11.8));
////	double bn = 0.025*exp(-tempV / 22.222);
////	m_p = Iterative(m_p, an, bn, TimeStep);
////	return m_p;
////}
////__device__ double CKv::Compute_KI(double tempV, double TimeStep)
////{
////	double tmp_n = Compute_n(tempV, TimeStep);
////	double K_I = g_Kv * pow(tmp_n, 2)*(tempV + 77);
////	return K_I;
////}
////__device__ CKv::CKv()
////{}
////__device__ CKv::~CKv()
////{
////}
////
//////////////////////////////////////////////////////////Na
////
////class CNa
////{
////protected:
////	double g_Na;
////	double m_Nam;
////	double m_Nah;
////public:
////	__device__ CNa(double gna, double m, double h);
////	__device__ double Compute_m(double tempV, double TimeStep);
////	__device__ double Compute_h(double tempV, double TimeStep);
////	__device__ double Compute_NaI(double tempV, double TimeStep);
////	__device__ double Iterative(double tmp_n, double an, double bn, double TimeStep);
////	__device__ CNa();
////	__device__ ~CNa();
////};
////
////__device__ CNa::CNa(double gna, double m, double h)
////{
////	g_Na = gna;
////	m_Nam = m;
////	m_Nah = h;
////}
////__device__ double CNa::Compute_m(double tempV, double TimeStep)
////{
////	double am = 0.1*(tempV + 40) / (1 - exp(-0.1*(tempV + 40)));
////	double bm = 4 * exp(-(tempV + 65) *1.0 / 18);//////////18
////	m_Nam = Iterative(m_Nam, am, bm, TimeStep);
////	return m_Nam;
////}
////__device__ double CNa::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double CNa::Compute_h(double tempV, double TimeStep)
////{
////	double ah = 0.07*exp(-(tempV + 65)*1.0 / 20);//////////20
////	double bh = 1.0 / (exp(-0.1*(tempV + 35)) + 1);
////	m_Nah = Iterative(m_Nah, ah, bh, TimeStep);
////	return m_Nah;
////}
////__device__ double CNa::Compute_NaI(double tempV, double TimeStep)
////{
////	double tmp_m = Compute_m(tempV, TimeStep);
////	double tmp_h = Compute_h(tempV, TimeStep);
////	double Na_I = g_Na * pow(tmp_m, 3)*tmp_h*(tempV - 55);
////	return Na_I;
////}
////__device__ CNa::CNa()
////{}
////__device__ CNa::~CNa()
////{}
////
////////////////////////////////////////////////////////Cell
////class CCell
////{
////protected:
////	double m_start;
////	double m_TimeStep;
////public:
////	CNa CNa_a;
////	CK CK_a;
////	CKM CKM_a;
////	CKv CKv_a;
////	CCa CCa_a;
////	float *m_pArray_V;
////
////public:
////	__device__ CCell(double start, double TimeStep, CNa Na_a, CK K_a, CKM KM_a, CKv Kv_a, CCa Ca_a);
////	__device__ double CCell::Iterative(double tmp_n, double an, double bn, double TimeStep);
////
////	__device__ double  CCell::GetStart();
////	__device__ double  CCell::GetTimeStep();
////	//CStringArray CCell::GetPArray_V();
////
////	__device__ ~CCell();
////};
////
////__device__ CCell::CCell(double start, double TimeStep, CNa Na_a, CK K_a, CKM KM_a, CKv Kv_a, CCa Ca_a)
////{
////	m_start = start;
////	CNa_a = Na_a;
////	CK_a = K_a;
////	CKM_a = KM_a;
////	CKv_a = Kv_a;
////	CCa_a = Ca_a;
////	//m_pArray_V = pArray_V;
////	m_TimeStep = TimeStep;
////}
////__device__ double CCell::Iterative(double tmp_n, double an, double bn, double TimeStep)
////{
////	double y_pre = tmp_n + (an*(1 - tmp_n) - bn*tmp_n)*TimeStep;
////	double y_next = tmp_n + (an*(1 - y_pre) - bn*y_pre)*TimeStep;;
////	double tmp_avgn = (y_pre + y_next)*1.0 / 2;
////	return tmp_avgn;
////}
////__device__ double  CCell::GetStart()
////{
////	double start = m_start;
////	return start;
////}
////
////__device__ double  CCell::GetTimeStep()
////{
////	double 	 TimeStep = m_TimeStep;
////	return TimeStep;
////}
////
////
////
////__device__ CCell::~CCell()
////{
////}
////
///////////////////////////////////////////////////////
////__device__ void Calculation_I(CCell Cell_B, float* pCurrent, double Mtime, int FlagParameter[], float gL, float C, float *pArrayA)
////{
////
////	double temp, y_preB, y_nextB;
////	double cell_B_K_I;
////	double cell_B_Na_I;
////	double cell_B_Ca_I;
////	double cell_B_KM_I;
////	double cell_B_Kv_I;
////	//double cell_pre_Nmda_a = Cell_pre.CNmda_a.CNMDA_I(tempV_a, tempVB, Time, Ta, Tb, Times_b, Cell_B.GetTimeStep());
////	///////
////	//////////////////////////////////
////	//Cell_B.m_pArray_V = new float[int(Mtime) * 100];
////	double tempVB = Cell_B.GetStart();
////	for (int k = 0; k < Mtime / Cell_B.GetTimeStep(); k++)
////	{
////		if (FlagParameter[1] == 0)
////			cell_B_K_I = 0;
////		else
////			cell_B_K_I = Cell_B.CK_a.Compute_KI(tempVB, Cell_B.GetTimeStep());
////		if (FlagParameter[0] == 0)
////			cell_B_Na_I = 0;
////		else
////			cell_B_Na_I = Cell_B.CNa_a.Compute_NaI(tempVB, Cell_B.GetTimeStep());
////		if (FlagParameter[4] == 0)
////			cell_B_Ca_I = 0;
////		else
////			cell_B_Ca_I = Cell_B.CCa_a.Compute_KI(tempVB, Cell_B.GetTimeStep());
////		if (FlagParameter[2] == 0)
////			cell_B_KM_I = 0;
////		else
////			cell_B_KM_I = Cell_B.CKM_a.Compute_KI(tempVB, Cell_B.GetTimeStep());
////		if (FlagParameter[3] == 0)
////			cell_B_Kv_I = 0;
////		else
////			cell_B_Kv_I = Cell_B.CKv_a.Compute_KI(tempVB, Cell_B.GetTimeStep());
////
////		y_preB = tempVB + Cell_B.GetTimeStep()*(gL*(Cell_B.GetStart() - tempVB) + pCurrent[k]
////			- cell_B_Na_I - cell_B_K_I - cell_B_Ca_I - cell_B_KM_I - cell_B_Kv_I) / C;
////
////		cell_B_K_I = Cell_B.CK_a.Compute_KI(y_preB, Cell_B.GetTimeStep());
////
////		cell_B_Na_I = Cell_B.CNa_a.Compute_NaI(y_preB, Cell_B.GetTimeStep());
////		cell_B_Ca_I = Cell_B.CCa_a.Compute_KI(y_preB, Cell_B.GetTimeStep());
////		cell_B_KM_I = Cell_B.CKM_a.Compute_KI(y_preB, Cell_B.GetTimeStep());
////		cell_B_Kv_I = Cell_B.CKv_a.Compute_KI(y_preB, Cell_B.GetTimeStep());
////
////		y_nextB = tempVB + Cell_B.GetTimeStep()*(gL*(Cell_B.GetStart() - y_preB) + pCurrent[k]
////			- cell_B_Na_I - cell_B_K_I - cell_B_Ca_I - cell_B_KM_I - cell_B_Kv_I) / C;
////
////		temp = (y_preB + y_nextB)*1.0 / 2;
////		pArrayA[k] = temp;
////		tempVB = temp;
////	}
////}
////
////
////__global__ void HH_EntranceA(float *population, int POPULATION_SIZE, double Mtime, double tempVB, double TimeStep, float* pCurrent, int FlagParameter[], float gL, float C, float *pArrayA, int VAR_NUMBER)
////{
////	//freopen("V_output.txt", "w", stdout);
////	//double Mtime = 100;
////	//double tempVB = -69;
////	//double TimeStep = 0.01;
////	//printf("v_%f ", tempVB);
////	int spike_length = int(Mtime / TimeStep);
////	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
////	//const int tid = threadIdx.x;
////	//if (gid < POPULATION_SIZE)
////	//{
////	//printf("gid_%d \n", gid);
////	float gNa = population[gid*VAR_NUMBER + 0];
////	float gK = population[gid*VAR_NUMBER + 1];
////	float gKM = population[gid*VAR_NUMBER + 2];
////	float gKv = population[gid*VAR_NUMBER + 3];
////	float gCa = population[gid*VAR_NUMBER + 4];
////	//printf("na_%f ", gNa);
////	CK K_a(gK, 0.318);//36
////	CNa Na_a(gNa, 0.053, 0.596);///(120, 0.053, 0.596)
////	CKM KM_a(gKM, 0);///(120, 0.053, 0.596)
////	CKv Kv_a(gKv, 0);///(120, 0.053, 0.596)
////	CCa Ca_a(gCa, 0);///(120, 0.053, 0.596)
////	//double m_I = 10;
////	//printf("s_%d ", 2);
////	CCell Cell_D(tempVB, TimeStep, Na_a, K_a, KM_a, Kv_a, Ca_a);
////	Calculation_I(Cell_D, pCurrent, Mtime, FlagParameter, gL, C, &pArrayA[gid*spike_length]);
////	//}
////	__syncthreads();
////	//for (int i = 0; i < int(Mtime) * 100; i++)
////	//{
////	//	printf("%f\n", pArrayA[i]);
////	//}
////	////////////////
////	//delete pArrayA;
////
////}
////void HH_Entrance(float *population, double Mtime, double tempVB, double TimeStep, float* m_I, int FlagParameter[], float gL, float C, float *pArrayA, int POPULATION_SIZE)
////{
////	int VAR_NUMBER = 5;
////	float *devicePopulation = 0;
////	float *deviceM_I = 0;
////	float *devicepArrayA = 0;
////	int *deviceFlagParameter = 0;
////	const int THREADS_PER_BLOCK = 1024;
////	const int BLOCKS_NUMBER = (POPULATION_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
////	int spike_length = int(Mtime / TimeStep);
////	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
////	cudasafe(hipMalloc((void **)&deviceM_I, spike_length * sizeof(float)), "Could not allocate memory for deviceM_I");
////
////	cudasafe(hipMalloc((void **)&devicepArrayA, POPULATION_SIZE * spike_length * sizeof(float)), "Could not allocate memory for devicepArrayA");
////	cudasafe(hipMalloc((void **)&deviceFlagParameter, VAR_NUMBER* sizeof(int)), "Could not allocate memory for deviceFlagParameter");
////
////	//cudasafe(hipMemcpy(devicepArrayA, pArrayA, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy pArrayA to device");
////	cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");
////	cudasafe(hipMemcpy(deviceM_I, m_I, spike_length* sizeof(float), hipMemcpyHostToDevice), "Could not copy m_I to device");
////
////	cudasafe(hipMemcpy(deviceFlagParameter, FlagParameter, VAR_NUMBER * sizeof(int), hipMemcpyHostToDevice), "Could not copy FlagParameter to device");
////
////	HH_EntranceA << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, POPULATION_SIZE, Mtime, tempVB, TimeStep, deviceM_I, deviceFlagParameter, gL, C, devicepArrayA, VAR_NUMBER);
////	cudasafe(hipMemcpy(pArrayA, devicepArrayA, POPULATION_SIZE * spike_length * sizeof(float), hipMemcpyDeviceToHost), "Could not copy pArrayA from device");
////
////	cudasafe(hipFree(devicepArrayA), "Could not free devicepArrayA");
////	cudasafe(hipFree(devicePopulation), "Could not free devicePopulation");
////	cudasafe(hipFree(deviceM_I), "Could not free deviceM_I");
////	cudasafe(hipFree(deviceFlagParameter), "Could not free deviceFlagParameter");
////}

////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////