#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <iomanip>
#include <fstream>  
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <time.h> 
#include <string>
#include "struct_cu.h"
#include "constants.h"
//#include "buffer.h"
using namespace std;
// assume block size equal population size

void cudasafe(hipError_t error, char* message = "Error occured") {
	if(error != hipSuccess) {
		fprintf(stderr,"ERROR: %s : %i\n", message, error);
		exit(-1);
	}
}

__global__ void randomInit(hiprandState* state, unsigned long seed) {
	int tid = threadIdx.x;
	hiprand_init(seed, tid, 0, state + tid);
}

__device__ float fitness(M_args deviceParameter, M_args_Tset deviceParameter_Tset,float tau) {
    float result = 0;
	//printf("%d_a",deviceParameter.spike_data_num);
	//printf("%d_b", deviceParameter_Tset.length);
	for (size_t i = 0; i<deviceParameter_Tset.length; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
	{
		result += expf(-fabsf(deviceParameter_Tset.spike_TestData[i] - deviceParameter_Tset.spike_TestData[j])*1.0 / tau);
		//printf("%f_3 ", result);
		// ++curPos;
	}

	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter.spike_data_num; ++j)
		{
			result += expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter.spike_data[j])*1.0 / tau);
			//printf("%f_2 ", result);
			// ++curPos;
		}
	for (size_t i = 0; i<deviceParameter.spike_data_num; ++i)
		for (size_t j = 0; j<deviceParameter_Tset.length; ++j)
		{
			//printf("%f_c ", deviceParameter.spike_data[i]);
			//printf("%f_d ", deviceParameter_Tset.spike_TestData[j]);
			result -= 2*expf(-fabsf(deviceParameter.spike_data[i] - deviceParameter_Tset.spike_TestData[j])*1.0 / tau);
			//printf("%f_1 ", result);
			// ++curPos;
		}
    //printf("%f_4 ", result);
    return result;
}


__global__ void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau, int genindex, int MaxGeneration, M_args_Bound *deviceParameter_Bound, const int POPULATION_SIZE, float crossver,float mutations) {
	// we first have to calculate the score for the first half of threads
	//const float *curPos = sharedPopulation[tid];
	__shared__ float sharedPopulation[THREADS_PER_BLOCK*2][VAR_NUMBER];
	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
	//__shared__ float **sharedPopulation = new float*[THREADS_PER_BLOCK * 2];
	__shared__ float sharedScore[THREADS_PER_BLOCK*2];
	//const float SIGN[2] = { -1.0f, 1.0f };
	//const float MULT[2] = { 1.0f, 0.0f };

	const int gid = blockDim.x * blockIdx.x + threadIdx.x + THREADS_PER_BLOCK;
	const int tid = threadIdx.x;
	printf("i_%f ", crossver);
	// loading initial random population into shared memory
	if ((gid + THREADS_PER_BLOCK)< POPULATION_SIZE) {
		for (int i = 0; i < VAR_NUMBER; ++i){
			__syncthreads();

			sharedPopulation[tid][i] = population[gid * VAR_NUMBER + i];
			sharedPopulation[tid + THREADS_PER_BLOCK][i] = population[(gid + THREADS_PER_BLOCK) * VAR_NUMBER + i];
		}
		M_args_Tset curPos = deviceParameter_Tset[gid];
		sharedScore[tid] = fitness(deviceParameter, curPos, tau);
		M_args_Tset curPos_b = deviceParameter_Tset[(gid + THREADS_PER_BLOCK)];
		sharedScore[tid + THREADS_PER_BLOCK] = fitness(deviceParameter, curPos_b, tau);
		__syncthreads();
		}

	//sharedScore[tid + THREADS_PER_BLOCK] = 123123.0;
	hiprandState &localState = randomStates[(tid*genindex)%THREADS_PER_BLOCK];
	__syncthreads();


	//__syncthreads();
	if (genindex <= MaxGeneration && (gid) < POPULATION_SIZE)
	{// selection
		// first half of threads writes best individual into its position
			if (sharedScore[tid] > sharedScore[tid + THREADS_PER_BLOCK]) {
				for (int i = 0; i < VAR_NUMBER; ++i)
					sharedPopulation[tid][i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
				sharedScore[tid] = sharedScore[tid + THREADS_PER_BLOCK];
			}
		
		__syncthreads();
		////int temp_size = THREADS_PER_BLOCK*(1 - crossver);
		////if (tid<temp_size && sharedScore[tid] > sharedScore[tid + temp_size]) {
		////	for (int i = 0; i < VAR_NUMBER; ++i)
		////		sharedPopulation[tid][i] = sharedPopulation[tid + temp_size][i];
		////	sharedScore[tid] = sharedScore[tid + temp_size];
		////}
		__syncthreads();
		// crossover
		////printf("i_%f ", crossver);
		////printf("gi_%f ", hiprand_uniform(&localState));
		if (hiprand_uniform(&localState) < crossver) {
			hiprandState &localState1 = randomStates[tid * 3];
			hiprandState &localState2 = randomStates[tid * 4];
			const int first = hiprand_uniform(&localState1) * THREADS_PER_BLOCK;
			const int second = hiprand_uniform(&localState2) * THREADS_PER_BLOCK;
			const float weight = hiprand_uniform(&localState1);
			for (int i = 0; i < VAR_NUMBER; ++i) {
				////const float Temp_weight = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
				//////printf("i_%f ", Temp_weight);
				////if (Temp_weight>deviceParameter_Bound[i].g[1] || Temp_weight < deviceParameter_Bound[i].g[0])
				////	sharedPopulation[tid + THREADS_PER_BLOCK][i] = deviceParameter_Bound[i].g[0] + weight*(deviceParameter_Bound[i].g[1] - deviceParameter_Bound[i].g[0]);
				////else
				////	sharedPopulation[tid + THREADS_PER_BLOCK][i] = Temp_weight;
				//printf("i_%f %f", deviceParameter_Bound[i].g[0], deviceParameter_Bound[i].g[1]);
				//printf("i_%f ", Temp_weight);
				sharedPopulation[tid + THREADS_PER_BLOCK][i] = sharedPopulation[first][i] * weight + sharedPopulation[second][i] * (1.0f - weight);
				
			}
		}
		__syncthreads();

		// mutations on second half of population
		hiprandState &localState3= randomStates[tid*5];
		if (hiprand_uniform(&localState3) < mutations) {
			//const float order = (hiprand_uniform(&localState) - mutations*1.0/2);
			float guass = hiprand_normal(&localState3);
			for (int i = 0; i < VAR_NUMBER; ++i) {
				////const float mult = MULT[order < 0.0f];
				////const float sign = SIGN[order < 0.0f];
				////sharedPopulation[tid + THREADS_PER_BLOCK][i] += powf(10.0f, order + order_deviation) * sign * mult;
				///sharedPopulation[tid + THREADS_PER_BLOCK][i] += sharedPopulation[tid + THREADS_PER_BLOCK][i] * order_deviation;
				////if (sharedPopulation[tid + THREADS_PER_BLOCK][i]>deviceParameter_Bound[i].g[1] || sharedPopulation[tid + THREADS_PER_BLOCK][i] < deviceParameter_Bound[i].g[0])
				sharedPopulation[tid + THREADS_PER_BLOCK][i] = (deviceParameter_Bound[i].g[0] + deviceParameter_Bound[i].g[1]) / 2 + guass;
			}
		}
		__syncthreads();
		const int third = hiprand_uniform(&localState)*THREADS_PER_BLOCK;
		//////sharing a part of population with others
		////if (hiprand_uniform(&localState)<0.1) {
		////	for (int i = 0; i < VAR_NUMBER; ++i)
		////		population[gid * VAR_NUMBER + i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
		////}

		// take some best individuals from neighbour
		if ((blockIdx.x + third) % 3 == 0) {
			if (hiprand_uniform(&localState) < 0.11) {
				const int anotherBlock = hiprand_uniform(&localState) * (POPULATION_SIZE / THREADS_PER_BLOCK);
				const int ngid = blockDim.x * anotherBlock + threadIdx.x;
				for (int i = 0; i < VAR_NUMBER; ++i)
					sharedPopulation[tid][i] = population[ngid * VAR_NUMBER + i];
				//sharedScore[tid] = fitness(sharedPopulation[tid], deviceParameter);
				//sharedScore[tid]=fitness(deviceParameter, curPos_b, tau);
			}
		}
	}
	__syncthreads();
	////// output current population back
	if ((gid ) < POPULATION_SIZE) {
		for (int i = 0; i < VAR_NUMBER; ++i)
			{
				population[gid * VAR_NUMBER + i] = sharedPopulation[tid][i];
				population[(gid + THREADS_PER_BLOCK) * VAR_NUMBER + i] = sharedPopulation[tid + THREADS_PER_BLOCK][i];
				printf("%f ", population[gid * VAR_NUMBER + i]);
			}
	
	if (genindex <= MaxGeneration)
		{
			score[gid].score = sharedScore[tid];
		}
	}
}


void printFinalPopulation(float* population, const ScoreWithId* deviceScore, const int POPULATION_SIZE) {
	////float **population = new float *[POPULATION_SIZE];
	////for (int i = 0; i < POPULATION_SIZE; i++)
	////	population[i] = new float[VAR_NUMBER];
	////cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

	ScoreWithId *score = new ScoreWithId[POPULATION_SIZE];
	cudasafe(hipMemcpy(score, deviceScore, POPULATION_SIZE * sizeof (ScoreWithId), hipMemcpyDeviceToHost), "Could not copy score to host");

	///std::cout.cetf(std::ios::fixed);
	std::cout.precision(12);

	for (int i = 0; i<1; ++i) {
		std::cout <<"gen_index"<< std::setw(15) << i << ' ';
	}
	std::cout << std::endl;

	for (int i=0; i<VAR_NUMBER; i++) {
			for (int u=0; u<POPULATION_SIZE; ++u) {
				std::cout << std::setw(15) << population[u*VAR_NUMBER + i] << ' ';
			}
			std::cout << std::endl;
		}
	std::cout << "Score: " << std::endl;
	for (int i = 0; i<POPULATION_SIZE; ++i) {
		std::cout << std::setw(15) << score[i].score << ' ';
	}
	std::cout << std::endl;
	//delete population;
}
extern "C" float solveGPU(M_args &Parameter_, double Mtime, double tempVB, double TimeStep, double m_I, int FlagParameter[], M_args_Bound Parameter_Bound[], int MaxGeneration, float gL, float C, const int POPULATION_SIZE, float crossver, float mutations, stringstream &strResult) {
    cudasafe(hipSetDevice(0), "Could not set device 0");
	float ans = 0;
	clock_t start, finish, startMy, finishMy;
	float tau = 12;
	//M_args *IndexParameter_ = new M_args[MaxGeneration];
	//IndexParameter_ = 0;
	//IndexParameter_[0] = Parameter_;
	/////////////////////////////////////
	////cout << Parameter_.spike_data_num << endl;
	////for (int i = 0; i < Parameter_.spike_data_num; i++)
	////	cout << Parameter_.spike_data[i] << " ";
	////cout << endl;
	////cout << mutations << " " << crossver<<endl;
	////std::cout << std::setw(15) << Parameter_Bound[0].g[0] << " " << Parameter_Bound[0].g[1] << " " << tempVB << endl;
	std::cout << FlagParameter[0] << ' ' << FlagParameter[1] << ' ' << FlagParameter[2] << ' ' << FlagParameter[3] << ' ' << FlagParameter[4] << endl;
	std::cout << Parameter_Bound[0].g[0] << ' ' << Parameter_Bound[1].g[0] << ' ' << Parameter_Bound[2].g[0] <<' ' << Parameter_Bound[3].g[0]<<endl;
	//////////////////////////////////////
	float *population = new float[POPULATION_SIZE*VAR_NUMBER];
	#pragma omp parallel for num_threads(4)
	for (int i=0; i<POPULATION_SIZE; i++) {
		for (int j=0; j<VAR_NUMBER; j++) {
			if (FlagParameter[j] == 0)
			{
				population[i*VAR_NUMBER + j] = 0;
			}
			else
			{
				population[i*VAR_NUMBER + j] = (float_random(Parameter_Bound[j]));
				//std::cout << std::setw(15) << population[i*VAR_NUMBER + j] << ' ';
			}
		}
		//std::cout << endl;
	}

	M_args_Tset *Parameter_Tset=new M_args_Tset[POPULATION_SIZE];

	// copying population to device
	float *devicePopulation = 0;
	float *nextGeneration = 0;
	M_args_Tset *deviceParameter_Tset = 0;
	
	hiprandState* randomStates;
	M_args deviceParameter_;
	//deviceParameter_.current_data_num = Parameter_.current_data_num;
	deviceParameter_.spike_data_num = Parameter_.spike_data_num;
	M_args_Bound *deviceParameter_Bound = 0;
	//int DataLength = getArrayLen(Parameter_.spike_data);


	//int DataLengthC = getArrayLen(Parameter_.current_data);
	//Parameter_.length = DataLength;
	cudasafe(hipMalloc((void **)&deviceParameter_Bound, VAR_NUMBER * sizeof(M_args_Bound)), "Could not allocate memory for deviceParameter_Bound");

	cudasafe(hipMalloc(&randomStates, THREADS_PER_BLOCK * sizeof(hiprandState)), "Could not allocate memory for randomStates");
	cudasafe(hipMalloc((void **)&devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for devicePopulation");
	cudasafe(hipMalloc((void **)&nextGeneration, POPULATION_SIZE * VAR_NUMBER * sizeof(float)), "Could not allocate memory for nextGeneration");
	cudasafe(hipMalloc((void **)&deviceParameter_Tset, 2*POPULATION_SIZE * sizeof (M_args_Tset)), "Could not allocate memory for deviceParameter_Tset");
	//cudasafe(hipMalloc((void **)&deviceParameter_.current_data, Parameter_.current_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	cudasafe(hipMalloc((void **)&deviceParameter_.spike_data, Parameter_.spike_data_num*sizeof(float)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_, sizeof(M_args)), "Could not allocate memory for deviceParameter_");
	//cudasafe(hipMalloc((void **)&deviceParameter_.spike_TestData, DataLength*sizeof(float)), "Could not allocate memory for deviceParameter_");

	///cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");
	//cudasafe(hipMemcpy(deviceParameter_.current_data, Parameter_.current_data, Parameter_.current_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_current_data to device");
	cudasafe(hipMemcpy(deviceParameter_.spike_data, Parameter_.spike_data, Parameter_.spike_data_num*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_spike_data to device");

	//cudasafe(hipMemcpy(deviceParameter_.spike_TestData, Parameter_.spike_TestData, DataLength*sizeof(float), hipMemcpyHostToDevice), "Could not copy Parameter_ to device");
	for (int kb = 0; kb<VAR_NUMBER;kb++)
		cudasafe(hipMemcpy(&deviceParameter_Bound[kb], &Parameter_Bound[kb], 2 * sizeof(float),hipMemcpyHostToDevice), "Could not allocate memory for deviceParameter_Bound");


	// invoking random init
	randomInit<<<1, THREADS_PER_BLOCK>>>(randomStates, 900);
	cudasafe(hipGetLastError(), "Could not invoke kernel randomInit");
	cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling randomInit");

	const int BLOCKS_NUMBER = (POPULATION_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	
	//__shared__ float sharedPopulation[THREADS_PER_BLOCK * 2][VAR_NUMBER];
	//__shared__ float sharedScore[THREADS_PER_BLOCK * 2];
    //for (int i=0; i<1115; i++) {
	//void GAKernel_GenEach(float* population, ScoreWithId* score, hiprandState* randomStates, M_args deviceParameter, M_args_Tset *deviceParameter_Tset, float tau)
	for (int k = 1; k <=MaxGeneration; k++) 
	{
		startMy = clock();
		start = clock();
		hDll = LoadLibrary("CHdll.dll");
		int spike_length = int(Mtime / TimeStep);
		float *temp_data = new float[spike_length*POPULATION_SIZE];
		HH_return(population, VAR_NUMBER, Mtime, tempVB, TimeStep, m_I, FlagParameter, gL, C, temp_data,POPULATION_SIZE);
		//Parameter_Tset[j].spike_TestData = HH_SpikeTime(temp_data, Mtime, TimeStep, Parameter_Tset[j].length, Parameter_.spike_data_num);
		/////////////
		////for (int j = 0; j < POPULATION_SIZE; j++)
		////	for (int i = 0; i < spike_length; i++)
		////		cout << temp_data[j*spike_length+i] << endl;
			//////////////////
		//#pragma omp parallel for
		for (int j = 0; j < POPULATION_SIZE; j++)
		{
			float *temp_spike_TestData;
			vector<float>mdata_tmp = HH_SpikeTime(&temp_data[j*spike_length], Mtime, TimeStep, Parameter_Tset[j].length, Parameter_.spike_data_num);
			Parameter_Tset[j].spike_TestData = new float[mdata_tmp.size()];
			convert_data_two(mdata_tmp, Parameter_Tset[j].spike_TestData);
			hipMalloc(&temp_spike_TestData, Parameter_Tset[j].length*sizeof(float));
			//std::cout << Parameter_Tset[j].length << std::endl;
			cudasafe(hipMemcpy(&deviceParameter_Tset[j], &Parameter_Tset[j], sizeof (M_args_Tset), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset1 to device");
			cudasafe(hipMemcpy(temp_spike_TestData, Parameter_Tset[j].spike_TestData, (Parameter_Tset[j].length*sizeof(float)), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData2 to device");
			cudasafe(hipMemcpy(&deviceParameter_Tset[j].spike_TestData, &temp_spike_TestData, sizeof(float*), hipMemcpyHostToDevice), "Could not copy deviceParameter_Tset_spike_TestData to device");
			cudasafe(hipFree(temp_spike_TestData), "Could not free temp_spike_TestData");
			delete Parameter_Tset[j].spike_TestData;
			//cout << j << endl;
		}
		delete temp_data;
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time1 " << duration << endl;
		ScoreWithId *deviceScore = 0;
		cudasafe(hipMalloc((void **)&deviceScore, POPULATION_SIZE * sizeof (ScoreWithId)), "Could not allocate memory for deviceScore");
		cudasafe(hipMemcpy(devicePopulation, population, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyHostToDevice), "Could not copy population to device");

		start = clock();
		GAKernel_GenEach << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau, k, MaxGeneration, deviceParameter_Bound, POPULATION_SIZE, crossver, mutations);
		//delete[]population;
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time2 " << duration << endl;
		//float *population = new float[POPULATION_SIZE * VAR_NUMBER];
		cudasafe(hipMemcpy(population, devicePopulation, POPULATION_SIZE * VAR_NUMBER * sizeof(float), hipMemcpyDeviceToHost), "Could not copy population from device");

		//printf("%d_1111\n", k);
		//printFinalPopulation(population, deviceScore, POPULATION_SIZE);
		//printf("%d_2222\n", k);
		//while (MFlage)
		/////////////////////exit-code
		if (myexit)
		{
			cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
			cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");

			cudasafe(hipFree(randomStates), "Could not free randomStates");
			cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
			cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
			delete Parameter_Tset;
			delete population;
			return 0;
		}
		//////////////////////
		while (MFlage == 2){}
		MFlage = 1;
		start = clock();
		ans = FinallResult(k, tau, population, Mtime, tempVB, TimeStep, m_I, Parameter_, FlagParameter, gL, C, Parameter_Bound, POPULATION_SIZE, strResult);
		finish = clock();
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		cout << "time3 " << duration << endl;
		MFlage = 3;
		cudasafe(hipFree(deviceScore), "Failed to free deviceScore");
		FreeLibrary(hDll);
		finishMy = clock();
		durationMy = (double)(finishMy - startMy) / CLOCKS_PER_SEC;
		cout << "timeAll " << durationMy << endl;
		/////////////////////exit-code
		if (myexit)
		{
			cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");

			cudasafe(hipFree(randomStates), "Could not free randomStates");
			cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
			cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
			delete Parameter_Tset;
			delete population;
			return 0;
		}
		//////////////////////
		////GAKernel_gen << <BLOCKS_NUMBER, THREADS_PER_BLOCK >> >(devicePopulation, sharedPopulation, sharedScore, deviceScore, randomStates, deviceParameter_, deviceParameter_Tset, tau);
	}
	
	//ans = FinallResult(tau, population, Mtime, tempVB, TimeStep, m_I, Parameter_, FlagParameter, gL, C, Parameter_Bound, POPULATION_SIZE, strResult);
    cudasafe(hipGetLastError(), "Could not invoke GAKernel");
    cudasafe(hipDeviceSynchronize(), "Failed to syncrhonize device after calling GAKernel");

    //printPopulation(devicePopulation, deviceScore);
    //}

	// freeing memory
	cudasafe(hipFree(devicePopulation), "Failed to free devicePopulation");
	
	cudasafe(hipFree(randomStates), "Could not free randomStates");
	cudasafe(hipFree(nextGeneration), "Could not free nextGeneration");
	cudasafe(hipFree(deviceParameter_Tset), "Could not free deviceParameter_Tset");
	delete Parameter_Tset;
	delete population;
	myexit = true;
	return ans;
}


